#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <thrust/device_vector.h>

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_cooperative_groups.h>
//#include <cuda/barrier>
#include <vector>
#include <hip/hip_vector_types.h>

__global__ void maxmin_tensor(float* dst,float* src,int total,float* max,float* min_src) {
    int wch_max = blockIdx.x;
    float maxV = max_src[wch_max];
    float minV = min_src[wch_max];
    int tid =  blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > total) return;
    int offset = wch_max * total;
    for(;tid < total ;tid += blockDim.x * gridDim.x) {
        dst[offset + tid] = min(maxV,max(minV,src[offset + tid]));
    }

}

__global__ void maxmin_single(float* dst,float* src,int total,float* max_src,float* min_src) {
    float maxV = max_src[0];
    float minV = min_src[0];
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > total) return;
    for(;tid < total ;tid += blockDim.x * gridDim.x) {
        dst[tid] = min(maxV,max(minV,src[tid]));
    }
}

torch::Tensor maxmin_cuda_forward(
    torch::Tensor input,
    torch::Tensor max,
    torch::Tensor min) {
    const auto batch_size = input.size(0);
    const auto C  =input.size(1);
    const auto H  =input.size(2);
    const auto W  =input.size(3);

    auto dtype = input.type();
    auto dev = input.device();
    auto output = torch::empty({batch_size, C, H, W}, dtype);

    const auto maxSize = max.size(0);

    if (maxSize == 1) {
        const auto total = batch_size * C * H * W;
        dim3 grid(32);
        dim3 block(256);
        maxmin_single<<<grid,block>>>((float*)output.data_ptr(),(float*)input.data_ptr(),total,(float*)max.data_ptr(),(float*)min.data_ptr());
    } else {
        dim3 grid(batch_size);
        dim3 block(256);
        const auto total = C * H * W;
        maxmin_tensor<<<grid,block>>>((float*)output.data_ptr(),(float*)input.data_ptr(),total,(float*)max.data_ptr(),(float*)min.data_ptr());
    }
    return output;
}
