#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <thrust/device_vector.h>

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_cooperative_groups.h>
//#include <cuda/barrier>
#include <vector>
#include <hip/hip_vector_types.h>

__global__ void maxmin_tensor(float* dst,const float* src,int total,const float* min_src,const float* max_src) {
    int wch_max = blockIdx.x;
    float maxV = max_src[wch_max];
    float minV = min_src[wch_max];
    int tid =  threadIdx.x;
    if (tid >= total) return;
    const int offset = wch_max * total;
    for(;tid < total ;tid += gridDim.x) {
        dst[offset + tid] = min(maxV,max(minV,src[offset + tid]));
    }

}

__global__ void maxmin_single(float* dst,const float* src,const int total,const float* min_src,const float* max_src) {
    float maxV = max_src[0];
    float minV = min_src[0];
    int tid = threadIdx.x;
    if (tid > total) return;
    for(;tid < total ;tid += blockDim.x) {
        dst[tid] = min(maxV,max(minV,src[tid]));
    }
}

torch::Tensor own_max_min_cuda(
    torch::Tensor input,

    torch::Tensor min,
    torch::Tensor max) {
    const auto batch_size = input.size(0);
    const auto elements  = input.numel() / batch_size;

    auto dtype = input.type();
    auto dev = input.device();
    auto output = input.new_empty(input.sizes()).to(dtype);

    const auto maxSize = max.size(0);

    if (maxSize == 1) {
        const auto total = batch_size * elements;
        dim3 grid(1);
        dim3 block(256);
        maxmin_single<<<grid,block>>>((float*)output.data_ptr(),(float*)input.data_ptr(),total,(float*)min.data_ptr(),(float*)max.data_ptr());
    } else {
        dim3 grid(batch_size);
        dim3 block(256);
        const auto total = elements;
        maxmin_tensor<<<grid,block>>>((float*)output.data_ptr(),(float*)input.data_ptr(),total,(float*)min.data_ptr(),(float*)max.data_ptr());
    }
    return output;
}
